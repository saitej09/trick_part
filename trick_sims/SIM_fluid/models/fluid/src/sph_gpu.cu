#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include "sph_gpu.h"


#define NUM_THREADS 1024

Particle* d_particles;
Particle** d_spatial_grid;
int* d_cell_counts;
Fluid* d_fluid;
int* d_n;




__global__ void computeDensityAndPressureGPU(Particle* particles, int* n, Fluid* fluid) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;

	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;

	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		pi.rho = 0;
		
		for (int j = 0; j < *n; j++) {
			
			//Particle &pj = candidate_neighbors[j];
			
			Particle& pj = particles[j];
			float rij[3] = {pj.pos[0] - pi.pos[0], pj.pos[1] - pi.pos[1], pj.pos[2] - pi.pos[2]};
			float r = sqrt(rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2]);
			if (r >= 0 && r <= fluid->H) {
				pi.rho += fluid->MASS * fluid->POLY6 * pow(fluid->HSQ - r * r, 3.f);
				
			}
		}
/*
		int grid_x = fluid->CELLS_PER_DIM * ((pi.pos[0] + fluid->BOUND) / (2 * fluid->BOUND));
		int grid_y = fluid->CELLS_PER_DIM * ((pi.pos[1] + fluid->BOUND) / (2 * fluid->BOUND));
		int grid_z = fluid->CELLS_PER_DIM * ((pi.pos[2] + fluid->BOUND) / (2 * fluid->BOUND));

		
		
		for (int x = grid_x - 1; x <= grid_x + 1; x++) {
			for (int y = grid_y - 1; y <= grid_y + 1; y++) {
				for (int z = grid_z - 1; z <= grid_z + 1; z++) {
					
					if (x >= 0 && x < fluid->CELLS_PER_DIM && y >= 0 && y < fluid->CELLS_PER_DIM && z >= 0 && z < fluid->CELLS_PER_DIM) {
						int grid_idx = x + y * fluid->CELLS_PER_DIM + z * fluid->CELLS_PER_DIM * fluid->CELLS_PER_DIM;
						int num_neighbors = cell_counts[grid_idx];
						if (num_neighbors == 0) {
							continue;
						}
						Particle* candidate_neighbors = spatial_grid[grid_idx];
						
						
						for (int j = 0; j < num_neighbors; j++) {
							
							Particle &pj = candidate_neighbors[j];
							
							//Particle& pj = particles[j];
							float rij[3] = {pj.pos[0] - pi.pos[0], pj.pos[1] - pi.pos[1], pj.pos[2] - pi.pos[2]};
							float r = sqrt(rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2]);
							if (r >= 0 && r <= fluid->H) {
								pi.rho += fluid->MASS * fluid->POLY6 * pow(fluid->HSQ - r * r, 3.f);
								
							}
						}
						
					}
				}
			}
		}
*/

		pi.pressure = fluid->GAS_CONST * (pi.rho - fluid->REST_DENS);
	}

	if (*n % NUM_THREADS != 0 && tid < (*n - NUM_THREADS * block_size)) {
		int leftover_start = (*n / NUM_THREADS) * NUM_THREADS;
		p_start = tid + leftover_start;
		p_end = (tid + 1) + leftover_start;

		for (int i = p_start; i < p_end; i++) {
			Particle& pi = particles[i];
			pi.rho = 0;
			
			for (int j = 0; j < *n; j++) {
				
				//Particle &pj = candidate_neighbors[j];
				
				Particle& pj = particles[j];
				float rij[3] = {pj.pos[0] - pi.pos[0], pj.pos[1] - pi.pos[1], pj.pos[2] - pi.pos[2]};
				float r = sqrt(rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2]);
				if (r >= 0 && r <= fluid->H) {
					pi.rho += fluid->MASS * fluid->POLY6 * pow(fluid->HSQ - r * r, 3.f);
					
				}
			}

			pi.pressure = fluid->GAS_CONST * (pi.rho - fluid->REST_DENS);
		}
		
	}
}



__global__ void computeForcesGPU(Particle* particles, int* n, Fluid* fluid) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;

	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;

	//for(auto &pi : particles) {
	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		float pressure_force[3] = {0, 0, 0};
		float viscosity_force[3] = {0, 0, 0};
		//Particle* candidate_neighbors = all_neighbors[i];
		for (int j = 0; j < *n; j++) {
			//Particle& pj = candidate_neighbors[j];
			Particle& pj = particles[j];
			if (&pi != &pj) {
				float rij[3] = {pj.pos[0] - pi.pos[0], pj.pos[1] - pi.pos[1], pj.pos[2] - pi.pos[2]};
				float r = std::sqrt(rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2]);
				float rij_hat[3] = {rij[0] / r, rij[1] / r, rij[2] / r};
				if (r > 0 && r <= fluid->H) {
					pressure_force[0] -= rij_hat[0] * fluid->MASS * (pi.pressure + pj.pressure) / (2 * pj.rho) * fluid->SPIKY_GRAD * pow(fluid->H - r, 2.f);
					pressure_force[1] -= rij_hat[1] * fluid->MASS * (pi.pressure + pj.pressure) / (2 * pj.rho) * fluid->SPIKY_GRAD * pow(fluid->H - r, 2.f);
					pressure_force[2] -= rij_hat[2] * fluid->MASS * (pi.pressure + pj.pressure) / (2 * pj.rho) * fluid->SPIKY_GRAD * pow(fluid->H - r, 2.f);

					viscosity_force[0] += fluid->VISC * fluid->MASS * ((pj.velocity[0] - pi.velocity[0]) / pj.rho) * fluid->VISC_LAP * (fluid->H - r);
					viscosity_force[1] += fluid->VISC * fluid->MASS * ((pj.velocity[1] - pi.velocity[1]) / pj.rho) * fluid->VISC_LAP * (fluid->H - r);
					viscosity_force[2] += fluid->VISC * fluid->MASS * ((pj.velocity[2] - pi.velocity[2]) / pj.rho) * fluid->VISC_LAP * (fluid->H - r);


				}
			}
		}
		float G[3] = {0.f, fluid->G_STRENGTH * -9.8f, 0.f};
		float gravity_force[3] = {pi.rho * G[0], pi.rho * G[1], pi.rho * G[2]};
		pi.force[0] = viscosity_force[0] + pressure_force[0] + gravity_force[0];
		pi.force[1] = viscosity_force[1] + pressure_force[1] + gravity_force[1];
		pi.force[2] = viscosity_force[2] + pressure_force[2] + gravity_force[2];
	}

	if (*n % NUM_THREADS != 0 && tid < (*n - NUM_THREADS * block_size)) {
		int leftover_start = (*n / NUM_THREADS) * NUM_THREADS;
		p_start = tid + leftover_start;
		p_end = (tid + 1) + leftover_start;

		//for(auto &pi : particles) {
		for (int i = p_start; i < p_end; i++) {
			Particle& pi = particles[i];
			float pressure_force[3] = {0, 0, 0};
			float viscosity_force[3] = {0, 0, 0};
			//Particle* candidate_neighbors = all_neighbors[i];
			for (int j = 0; j < *n; j++) {
				//Particle& pj = candidate_neighbors[j];
				Particle& pj = particles[j];
				if (&pi != &pj) {
					float rij[3] = {pj.pos[0] - pi.pos[0], pj.pos[1] - pi.pos[1], pj.pos[2] - pi.pos[2]};
					float r = std::sqrt(rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2]);
					float rij_hat[3] = {rij[0] / r, rij[1] / r, rij[2] / r};
					if (r > 0 && r <= fluid->H) {
						pressure_force[0] -= rij_hat[0] * fluid->MASS * (pi.pressure + pj.pressure) / (2 * pj.rho) * fluid->SPIKY_GRAD * pow(fluid->H - r, 2.f);
						pressure_force[1] -= rij_hat[1] * fluid->MASS * (pi.pressure + pj.pressure) / (2 * pj.rho) * fluid->SPIKY_GRAD * pow(fluid->H - r, 2.f);
						pressure_force[2] -= rij_hat[2] * fluid->MASS * (pi.pressure + pj.pressure) / (2 * pj.rho) * fluid->SPIKY_GRAD * pow(fluid->H - r, 2.f);

						viscosity_force[0] += fluid->VISC * fluid->MASS * ((pj.velocity[0] - pi.velocity[0]) / pj.rho) * fluid->VISC_LAP * (fluid->H - r);
						viscosity_force[1] += fluid->VISC * fluid->MASS * ((pj.velocity[1] - pi.velocity[1]) / pj.rho) * fluid->VISC_LAP * (fluid->H - r);
						viscosity_force[2] += fluid->VISC * fluid->MASS * ((pj.velocity[2] - pi.velocity[2]) / pj.rho) * fluid->VISC_LAP * (fluid->H - r);


					}
				}
			}
			float G[3] = {0.f, fluid->G_STRENGTH * -9.8f, 0.f};
			float gravity_force[3] = {pi.rho * G[0], pi.rho * G[1], pi.rho * G[2]};
			pi.force[0] = viscosity_force[0] + pressure_force[0] + gravity_force[0];
			pi.force[1] = viscosity_force[1] + pressure_force[1] + gravity_force[1];
			pi.force[2] = viscosity_force[2] + pressure_force[2] + gravity_force[2];
		}
	}
}

__global__ void verletUpdatePosition(Particle* particles, int* n, Fluid* fluid) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;
	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		pi.pos[0] += fluid->DT * pi.velocity[0];
		pi.pos[1] += fluid->DT * pi.velocity[1];
		pi.pos[2] += fluid->DT * pi.velocity[2];
	}

	if (*n % NUM_THREADS != 0 && tid < (*n - NUM_THREADS * block_size)) {
		int leftover_start = (*n / NUM_THREADS) * NUM_THREADS;
		p_start = tid + leftover_start;
		p_end = (tid + 1) + leftover_start;
		for (int i = p_start; i < p_end; i++) {
			Particle& pi = particles[i];
			pi.pos[0] += fluid->DT * pi.velocity[0];
			pi.pos[1] += fluid->DT * pi.velocity[1];
			pi.pos[2] += fluid->DT * pi.velocity[2];
		}
	}
	
}

__global__ void timeIntegrationGPU(Particle* particles, int* n, Fluid* fluid) {
	int tid = threadIdx.x;
	// assuming n is a multiple of NUM_THREADS
	int block_size = *n / NUM_THREADS;
	int p_start = tid * block_size;
	int p_end = (tid + 1) * block_size;
	for (int i = p_start; i < p_end; i++) {
		Particle& pi = particles[i];
		//pi.pos += DT * pi.velocity;
		pi.velocity[0] += fluid->DT * pi.force[0] / pi.rho;
		pi.velocity[1] += fluid->DT * pi.force[1] / pi.rho;
		pi.velocity[2] += fluid->DT * pi.force[2] / pi.rho;

		if (pi.pos[2] - fluid->EPS < -fluid->BOUND) {
			pi.velocity[2] *= fluid->BOUND_DAMPING;
			pi.pos[2] = -fluid->BOUND + fluid->EPS;
		}

		if (pi.pos[2] + fluid->EPS > fluid->BOUND) {
			pi.velocity[2] *= fluid->BOUND_DAMPING;
			pi.pos[2] = fluid->BOUND - fluid->EPS;
		}

		if (pi.pos[1] - fluid->EPS < -fluid->BOUND) {
			pi.velocity[1] *= fluid->BOUND_DAMPING;
			pi.pos[1] = -fluid->BOUND + fluid->EPS;
		}

		if (pi.pos[1] + fluid->EPS > fluid->BOUND) {
			pi.velocity[1] *= fluid->BOUND_DAMPING;
			pi.pos[1] = fluid->BOUND - fluid->EPS;
		}

		if (pi.pos[0] - fluid->EPS < -fluid->BOUND) {
			pi.velocity[0] *= fluid->BOUND_DAMPING;
			pi.pos[0] = -fluid->BOUND + fluid->EPS;
		}
		if (pi.pos[0] + fluid->EPS > fluid->BOUND) {
			pi.velocity[0] *= fluid->BOUND_DAMPING;
			pi.pos[0] = fluid->BOUND - fluid->EPS;
		}
	}

	if (*n % NUM_THREADS != 0 && tid < (*n - NUM_THREADS * block_size)) {
		int leftover_start = (*n / NUM_THREADS) * NUM_THREADS;
		p_start = tid + leftover_start;
		p_end = (tid + 1) + leftover_start;

		for (int i = p_start; i < p_end; i++) {
			Particle& pi = particles[i];
			//pi.pos += DT * pi.velocity;
			pi.velocity[0] += fluid->DT * pi.force[0] / pi.rho;
			pi.velocity[1] += fluid->DT * pi.force[1] / pi.rho;
			pi.velocity[2] += fluid->DT * pi.force[2] / pi.rho;

			if (pi.pos[2] - fluid->EPS < -fluid->BOUND) {
				pi.velocity[2] *= fluid->BOUND_DAMPING;
				pi.pos[2] = -fluid->BOUND + fluid->EPS;
			}

			if (pi.pos[2] + fluid->EPS > fluid->BOUND) {
				pi.velocity[2] *= fluid->BOUND_DAMPING;
				pi.pos[2] = fluid->BOUND - fluid->EPS;
			}

			if (pi.pos[1] - fluid->EPS < -fluid->BOUND) {
				pi.velocity[1] *= fluid->BOUND_DAMPING;
				pi.pos[1] = -fluid->BOUND + fluid->EPS;
			}

			if (pi.pos[1] + fluid->EPS > fluid->BOUND) {
				pi.velocity[1] *= fluid->BOUND_DAMPING;
				pi.pos[1] = fluid->BOUND - fluid->EPS;
			}

			if (pi.pos[0] - fluid->EPS < -fluid->BOUND) {
				pi.velocity[0] *= fluid->BOUND_DAMPING;
				pi.pos[0] = -fluid->BOUND + fluid->EPS;
			}
			if (pi.pos[0] + fluid->EPS > fluid->BOUND) {
				pi.velocity[0] *= fluid->BOUND_DAMPING;
				pi.pos[0] = fluid->BOUND - fluid->EPS;
			}
		}
	}
}

void initSPH_GPU(std::vector<Particle>& particles, Fluid* fluid) {
	int n = fluid->NUM_PARTICLES;
	
	hipMalloc(&d_particles, n * sizeof(Particle));
	hipMalloc(&d_fluid, sizeof(Fluid));
	hipMalloc(&d_n, sizeof(int));

	hipMemcpy(d_fluid, fluid, sizeof(Fluid), hipMemcpyHostToDevice);
	hipMemcpy(d_particles, particles.data(), n * sizeof(Particle), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
		

	/*
		int num_cells = std::pow(fluid->CELLS_PER_DIM, 3);
		hipMallocManaged(&d_spatial_grid, num_cells * sizeof(Particle*));
		hipMallocManaged(&d_cell_counts, num_cells * sizeof(int));
		
		for (int i = 0; i < num_cells; i++) 
		{
			if (fluid->spatialGrid.find(i) != fluid->spatialGrid.end()) {
				
				std::vector<Particle> cellParticles = fluid->spatialGrid[i];
				
				Particle* array;
				hipMallocManaged(&array, cellParticles.size() * sizeof(Particle));
				d_cell_counts[i] = cellParticles.size();

				for(int j = 0; j < cellParticles.size(); j++) {
					array[j] = cellParticles[j];
				}


				d_spatial_grid[i] = array;
			} else {
				hipMalloc(&d_spatial_grid[i], sizeof(Particle));
				d_cell_counts[i] = n;
			}
		}
	} else {
		for (int i = 0; i < num_cells; i++) 
		{
			if (fluid->spatialGrid.find(i) != fluid->spatialGrid.end()) {
				
				std::vector<Particle> cellParticles = fluid->spatialGrid[i];
				
				Particle* array;
				hipMallocManaged(&array, cellParticles.size() * sizeof(Particle));
				d_cell_counts[i] = cellParticles.size();

				for(int j = 0; j < cellParticles.size(); j++) {
					array[j] = cellParticles[j];
				}

				hipFree(d_spatial_grid[i]);
				d_spatial_grid[i] = array;
			} else {
				d_cell_counts[i] = n;
			}
		}*/
	
}

void updateSPH_GPU(std::vector<Particle>& particles, Fluid* fluid) {
	int n = fluid->NUM_PARTICLES;

	verletUpdatePosition<<<1, NUM_THREADS>>>(d_particles, d_n, d_fluid);

	computeDensityAndPressureGPU << <1, NUM_THREADS >> > (d_particles, d_n, d_fluid);
	
	computeForcesGPU << <1, NUM_THREADS >> > (d_particles, d_n, d_fluid);
	
	timeIntegrationGPU << <1, NUM_THREADS >> > (d_particles, d_n, d_fluid);

	hipDeviceSynchronize();

	hipMemcpy(particles.data(), d_particles, n * sizeof(Particle), hipMemcpyDeviceToHost);
/*
	for (int i = 0; i < num_cells; i++) 
	{
		hipFree(&d_spatial_grid[i]);
			
	}
*/

	
	

}

void shutdownSPH_GPU(std::vector<Particle>& particles, Fluid* fluid) {
	hipFree(d_particles);
	hipFree(d_fluid);
	hipFree(d_n);
}
