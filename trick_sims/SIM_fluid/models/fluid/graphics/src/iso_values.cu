#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"

#include "grid_cell.h"
#include <stdio.h>


#define NUM_THREADS 1024



__global__ void updateIsoValuesGPU(GridCell* gridCells, float* particlePositions, int numParticles, int numCells, float radius) {
    int tid = threadIdx.x;

    int blockSize = numCells / NUM_THREADS;

    int g_start = tid * blockSize;
    int g_end = (tid + 1) * blockSize;
    
    for (int i = g_start; i < g_end; i++) {
        for (int j = 0; j < 8; j++) {
            
			gridCells[i].isoValues[j] = 0;

            glm::vec3 vertexPos(gridCells[i].vertices[j]);
            //printf("(%f, %f, %f): %d\n", vertexPos.x, vertexPos.y, vertexPos.z, tid);
            for (int k = 0; k < numParticles; k++) {
                float x = particlePositions[3*k];
                float y = particlePositions[3*k+1];
                float z = particlePositions[3*k+2];
                glm::vec3 particlePos(x, y, z);
                if (glm::length(vertexPos - particlePos) < radius) {
                    gridCells[i].isoValues[j]+=1;
                    
                }
            }
        }
    }
}


void updateIsoValuesGPUCaller(std::vector<GridCell>& gridCells, std::vector<float> particlePositions, float radius) {
	
    GridCell* d_gridCells;
    float* d_particlePositions;

    hipMalloc(&d_gridCells, gridCells.size() * sizeof(GridCell));
    hipMalloc(&d_particlePositions, particlePositions.size() * sizeof(float));

    hipMemcpy(d_gridCells, gridCells.data(), gridCells.size() * sizeof(GridCell), hipMemcpyHostToDevice);
    hipMemcpy(d_particlePositions, particlePositions.data(), particlePositions.size() * sizeof(float), hipMemcpyHostToDevice);
	
	

	updateIsoValuesGPU<<<1, NUM_THREADS>>>(d_gridCells, d_particlePositions, particlePositions.size() / 3, gridCells.size(), radius);


	hipDeviceSynchronize();

	hipMemcpy(gridCells.data(), d_gridCells, gridCells.size() * sizeof(GridCell), hipMemcpyDeviceToHost);

    hipFree(d_gridCells);
    hipFree(d_particlePositions);
}
